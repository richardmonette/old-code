#include "hip/hip_runtime.h"

// Date: Friday November 19th, 2010

// Description: GPU implementation of Jos Stam's GDC '03 paper "Real-time Fluid Dynamics For Games" using NVidia CUDA

// Author: Richard E. Monette

#include "../common/book.h"
#include "../common/gpu_anim.h"

#define DIM 512
#define DT 0.1f
#define DIFFUSION 0.0f
#define VISCOSITY 0.0f
#define FORCE 5.0f
#define SOURCE 10.0f

static float *u;
static float *v;
static float *u_prev;
static float *v_prev;
static float *dens;
static float *dens_prev;

texture<float> tex_ref;
texture<float> tex_ref2;
texture<float> tex_ref3;

static float *draw_buffer_dens;
static float *draw_buffer_u;
static float *draw_buffer_v;

static int win_id;
static int mouse_down[3];
static int omx, omy, mx, my;

dim3    blocks(DIM/16,DIM/16);
dim3    threads(16,16);

#define SWAP(x0,x) {float *tmp=x0;x0=x;x=tmp;}
#define IX(i,j) ((i)+(DIM)*(j))

__global__ void add_force(int pixel, float u_value, float v_value, float *uu, float *vv)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

	if (abs(offset - pixel) < 3)
	{
		uu[offset] = u_value;
		vv[offset] = v_value;
	}
}

__global__ void set_density(int pixel, float value, float *dens)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

	if (abs(offset - pixel) < 3)
	{
		dens[offset] = value;
	}
}

__global__ void add_source(float *t, float *s) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    t[offset] += DT * s[offset];
}

__global__ void clear_kernel(float *t) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    t[offset] = 0.0f;
}

__global__ void set_bnd_kernel_0(float *x)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i == 0) x[IX(  0, i  )] = tex1D(tex_ref, IX(1,i));
	if (i == DIM-1) x[IX(DIM-1, i  )] = tex1D(tex_ref, IX(DIM-2,i));

	if (j == 0) x[IX(  i, 0  )] = tex1D(tex_ref, IX(i,1));
	if (j == DIM-1) x[IX(  i, DIM-1)] = tex1D(tex_ref, IX(i,DIM-2));
}

__global__ void set_bnd_kernel_1(float *x)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i == 0) x[IX(  0, i  )] = -tex1D(tex_ref, IX(1,i));
	if (i == DIM-1) x[IX(DIM-1, i  )] = -tex1D(tex_ref, IX(DIM-2,i));
}

__global__ void set_bnd_kernel_2(float *x)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (j == 0) x[IX(  i, 0  )] = -tex1D(tex_ref, IX(i,1));
	if (j == DIM-1) x[IX(  i, DIM-1)] = -tex1D(tex_ref, IX(i,DIM-2));
}

__global__ void advect(float *d, float *d0, float *u, float *v) 
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = i + j * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (i == 0) left++;
	if (i == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (j == 0) top += DIM;
	if (j == DIM - 1) bottom -= DIM;

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1, dt0;

	dt0 = (float)DT * DIM;

	x = i - dt0 * tex1Dfetch(tex_ref2, offset);
	y = j - dt0 * tex1Dfetch(tex_ref3, offset);

	if (x < 0.5f) x = 0.5f;
	if (x > DIM + 0.5f) x = DIM + 0.5f;
	i0 = (int)x;
	i1 = i0 + 1;
	
	if (y < 0.5f) y = 0.5f;
	if (y > DIM + 0.5f) y = DIM + 0.5f;
	j0 = (int)y;
	j1 = j0 + 1;

	s1 = x - i0;
	s0 = 1 - s1;
	t1 = y - j0;
	t0 = 1 - t1;

	if (i0 < 0) i0 = 0;
	if (i1 > DIM - 1) i1 = DIM - 1;

	if (j0 < 0) j0 = 0;
	if (j1 > DIM - 1) j1 = DIM - 1;

	d[offset] = s0 * (t0 * tex1Dfetch(tex_ref, i0 + j0 * DIM) + t1 * tex1Dfetch(tex_ref, i0 + j1 * DIM)) + 
		        s1 * (t0 * tex1Dfetch(tex_ref, i1 + j0 * DIM) + t1 * tex1Dfetch(tex_ref, i1 + j1 * DIM));
}

__global__ void diffuse_kernel(float *x, float *x0) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = i + j * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (i == 0) left++;
	if (i == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (j == 0) top += DIM;
	if (j == DIM - 1) bottom -= DIM;

	const float a = (float)DT * (float)DIFFUSION * DIM * DIM;

	x[offset] = x0[offset] + a * (x[left] + x[right] + x[bottom] + x[top]) / (1 + 4 * a);
}

__global__ void project_kernel1(float *u, float *v, float *p, float *div, float h) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	div[IX(i,j)] = -0.5f*(u[IX(i+1,j)]-u[IX(i-1,j)]+v[IX(i,j+1)]-v[IX(i,j-1)])/DIM;
	p[IX(i,j)] = 0;
}

__global__ void project_kernel2(float * x, float * x0, float a, float c) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	x[IX(i,j)] = (  tex1Dfetch(tex_ref2, IX(i,j)) + a * (  tex1Dfetch(tex_ref, IX(i-1,j)) + tex1Dfetch(tex_ref, IX(i+1,j)) + tex1Dfetch(tex_ref, IX(i,j-1)) + tex1Dfetch(tex_ref, IX(i,j+1))   )  )/c;
}

__global__ void project_kernel3(float *u, float *v, float *p, float h) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

	u[IX(i,j)] -= 0.5f * DIM * (tex1Dfetch(tex_ref, IX(i+1,j)) - tex1Dfetch(tex_ref, IX(i-1,j)));
	
	v[IX(i,j)] -= 0.5f * DIM * (tex1Dfetch(tex_ref, IX(i,j+1)) - tex1Dfetch(tex_ref, IX(i,j-1)));
}

void diffuse(float *x, float *x0) 
{
	for (int k = 0; k < 20; k++)
	{
		diffuse_kernel<<<blocks, threads>>>(x, x0);
		
		hipBindTexture(NULL, tex_ref, x, DIM * DIM * sizeof(float));
		set_bnd_kernel_0<<<blocks, threads>>>(x);
	}
}

void project(float *u, float *v, float *p, float *div)
{
	float h = 1.0 / DIM;

	project_kernel1<<<blocks, threads>>>(u, v, p, div, h);

	hipBindTexture(NULL, tex_ref, div, DIM * DIM * sizeof(float));
	set_bnd_kernel_0<<<blocks, threads>>>(div); 

	hipBindTexture(NULL, tex_ref, p, DIM * DIM * sizeof(float));
	set_bnd_kernel_0<<<blocks, threads>>>(p);

	for (int k = 0; k < 20; k++)
	{
		hipBindTexture(NULL, tex_ref, p, DIM * DIM * sizeof(float));
		hipBindTexture(NULL, tex_ref2, div, DIM * DIM * sizeof(float));
		project_kernel2<<<blocks, threads>>>(p, div, 1, 4);

		hipBindTexture(NULL, tex_ref, p, DIM * DIM * sizeof(float));
		set_bnd_kernel_0<<<blocks, threads>>>(p);
	}

	hipBindTexture(NULL, tex_ref, p, DIM * DIM * sizeof(float));
	project_kernel3<<<blocks, threads>>>(u, v, p, h);

	hipBindTexture(NULL, tex_ref, u, DIM * DIM * sizeof(float));
	set_bnd_kernel_1<<<blocks, threads>>>(u);

	hipBindTexture(NULL, tex_ref, v, DIM * DIM * sizeof(float));
	set_bnd_kernel_2<<<blocks, threads>>>(v);
}

void dens_step(float *x, float *x0, float *u, float *v)
{
	add_source<<<blocks, threads>>>(x, x0);
	SWAP(x0, x);
	diffuse(x, x0);
	SWAP(x0, x);

	hipBindTexture(NULL, tex_ref, x0, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref2, u, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref3, v, DIM * DIM * sizeof(float));
	advect<<<blocks, threads>>>(x, x0, u, v);

	hipBindTexture(NULL, tex_ref, x, DIM * DIM * sizeof(float));
	set_bnd_kernel_0<<<blocks, threads>>>(x);
}

void vel_step(float *u, float *v, float *u0, float *v0)
{
	add_source<<<blocks, threads>>>(u, u0);
	add_source<<<blocks, threads>>>(v, v0);
	SWAP(u0, u);
	diffuse(u, u0);
	SWAP(v0, v);
	diffuse(v, v0);
	project(u, v, u0, v0);
	SWAP(u0, u);
	SWAP(v0, v);

	hipBindTexture(NULL, tex_ref, u0, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref2, u0, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref3, v0, DIM * DIM * sizeof(float));
	advect<<<blocks, threads>>>(u, u0, u0, v0);

	hipBindTexture(NULL, tex_ref, u, DIM * DIM * sizeof(float));
	set_bnd_kernel_1<<<blocks, threads>>>(u);

	hipBindTexture(NULL, tex_ref, v0, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref2, u0, DIM * DIM * sizeof(float));
	hipBindTexture(NULL, tex_ref3, v0, DIM * DIM * sizeof(float));
	advect<<<blocks, threads>>>(v, v0, u0, v0);

	hipBindTexture(NULL, tex_ref, v, DIM * DIM * sizeof(float));
	set_bnd_kernel_2<<<blocks, threads>>>(v);

	project(u, v, u0, v0);
}

static void get_from_UI ( float * dd, float * uu, float * vv )
{
	clear_kernel<<<blocks, threads>>>(uu);
	clear_kernel<<<blocks, threads>>>(vv);
	clear_kernel<<<blocks, threads>>>(dd);

	if ( !mouse_down[0] && !mouse_down[2] ) return;

	int i = (int)mx;
	int j = (int)DIM - my;

	if ( i < 1 || i > DIM || j < 1 || j > DIM ) return;

	if ( mouse_down[0] ) 
	{
		add_force<<<blocks, threads>>>(i + j * DIM, FORCE * (mx-omx), FORCE * (omy-my), uu, vv);
	}

	if ( mouse_down[2] ) 
	{
		set_density<<<blocks, threads>>>(i + j * DIM, SOURCE, dd);
	}

	omx = mx;
	omy = my;

	return;
}

void simulate() 
{
	get_from_UI ( dens_prev, u_prev, v_prev );
    vel_step(u, v, u_prev, v_prev);
	dens_step(dens, dens_prev, u, v);
}

static void pre_display ( void )
{
	glViewport( 0, 0, DIM, DIM );
	glMatrixMode( GL_PROJECTION );
	glLoadIdentity ();
	gluOrtho2D( 0.0, 1.0, 0.0, 1.0 );
	glClearColor( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear( GL_COLOR_BUFFER_BIT );
}

static void post_display ( void )
{
	glutSwapBuffers();
}

static void idle_func ( void )
{
	simulate();

	glutSetWindow(win_id);
	glutPostRedisplay();
}

void HSVtoRGB( float *r, float *g, float *b, float h, float s, float v )
{
	int i;
	float f, p, q, t;
	if( s == 0 ) 
	{
		*r = *g = *b = v;
		return;
	}
	h /= 60;
	i = floor( h );
	f = h - i;
	p = v * ( 1 - s );
	q = v * ( 1 - s * f );
	t = v * ( 1 - s * ( 1 - f ) );
	switch( i ) {
		case 0:
			*r = v;
			*g = t;
			*b = p;
			break;
		case 1:
			*r = q;
			*g = v;
			*b = p;
			break;
		case 2:
			*r = p;
			*g = v;
			*b = t;
			break;
		case 3:
			*r = p;
			*g = q;
			*b = v;
			break;
		case 4:
			*r = t;
			*g = p;
			*b = v;
			break;
		default:
			*r = v;
			*g = p;
			*b = q;
			break;
	}
}

static void display_func ( void )
{
	pre_display();

		hipMemcpy(draw_buffer_dens, dens, DIM * DIM * sizeof(float), hipMemcpyDeviceToHost);

		float d00, d01, d10, d11;
		float r, g, b;

		glBegin ( GL_QUADS );

			for (int i = 0; i < DIM; i++) 
			{
				for (int j = 0; j < DIM; j++) 
				{
					d00 = draw_buffer_dens[i + j * DIM];
					d01 = draw_buffer_dens[i + (j+1) * DIM];
					d10 = draw_buffer_dens[(i+1) + j * DIM];
					d11 = draw_buffer_dens[(i+1) + (j+1) * DIM];

					float x = i / DIM.f;
					float y = j / DIM.f;

					float h = 1.f / DIM.f;
					
					HSVtoRGB(&r, &g, &b, d00 * 360.f, 1, d00);
					glColor3f ( r, g, b ); glVertex2f ( x, y );
					
					HSVtoRGB(&r, &g, &b, d01 * 360.f, 1, d01);
					glColor3f ( r, g, b ); glVertex2f ( x+h, y );
					
					HSVtoRGB(&r, &g, &b, d10 * 360.f, 1, d10);
					glColor3f ( r, g, b ); glVertex2f ( x+h, y+h );
					
					HSVtoRGB(&r, &g, &b, d11 * 360.f, 1, d11);
					glColor3f ( r, g, b ); glVertex2f ( x, y+h );
				}
			}

		glEnd ();

	post_display();
}

static void reshape_func ( int width, int height )
{
	glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );
}

static void mouse_func ( int button, int state, int x, int y )
{
	omx = mx = x;
	omx = my = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void motion_func ( int x, int y )
{
	mx = x;
	my = y;
}

static void open_glut_window ( void )
{
	glutInitDisplayMode ( GLUT_RGBA | GLUT_DOUBLE );

	glutInitWindowPosition(0, 0);
	glutInitWindowSize(DIM, DIM);
	win_id = glutCreateWindow("CUDA Fluid Simulation");

	glClearColor( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers();
	glClear( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();

	pre_display();

	//glutKeyboardFunc ( key_func );
	glutMouseFunc ( mouse_func );
	glutMotionFunc ( motion_func );
	glutReshapeFunc ( reshape_func );
	glutIdleFunc ( idle_func );
	glutDisplayFunc ( display_func );
}

int main(int argc, char ** argv) 
{
	hipMalloc((void**)&u, DIM * DIM * sizeof(float));
	hipMalloc((void**)&v, DIM * DIM * sizeof(float));
	hipMalloc((void**)&u_prev, DIM * DIM * sizeof(float));
	hipMalloc((void**)&v_prev, DIM * DIM * sizeof(float));
	hipMalloc((void**)&dens, DIM * DIM * sizeof(float));
	hipMalloc((void**)&dens_prev, DIM * DIM * sizeof(float));

	draw_buffer_dens = (float*)malloc(DIM * DIM * sizeof(float));
	draw_buffer_u = (float*)malloc(DIM * DIM * sizeof(float));
	draw_buffer_v = (float*)malloc(DIM * DIM * sizeof(float));

	dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);

	clear_kernel<<<blocks, threads>>>(u);
	clear_kernel<<<blocks, threads>>>(v);
	clear_kernel<<<blocks, threads>>>(u_prev);
	clear_kernel<<<blocks, threads>>>(v_prev);
	clear_kernel<<<blocks, threads>>>(dens);
	clear_kernel<<<blocks, threads>>>(dens_prev);

	glutInit(&argc, argv);

	open_glut_window();

	glutMainLoop();

	hipFree(u);
	hipFree(v);
	hipFree(u_prev);
	hipFree(v_prev);
	hipFree(dens);
	hipFree(dens_prev);

	hipUnbindTexture(tex_ref);

	free(draw_buffer_dens);
	free(draw_buffer_u);
	free(draw_buffer_v);
}
